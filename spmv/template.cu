#include <stdio.h>
#include <chrono>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void
spmvCSRKernel(float *out, int *matCols, int *matRows, float *matData, float *vec, int dim)
{
    int outIdx   = blockDim.x * blockIdx.x + threadIdx.x;
    float outReg = 0;
    int start, end, i;
    if (outIdx < dim) {
        outReg = 0;
        start  = matRows[outIdx];
        end    = matRows[outIdx + 1];
        for (i = start; i < end; i++) {
            outReg += matData[i] * vec[matCols[i]];
        }
        out[outIdx] += outReg;
    }
}

__global__ void
spmvJDSKernel(float *out, int *matColStart, int *matCols, int *matRowPerm, int *matRows, float *matData, float *vec,
              int dim)
{
    int outIdx = blockDim.x * blockIdx.x + threadIdx.x;
    int prod, start, end, iterator;
    if (outIdx < dim) {
        prod  = 0;
        start = 0;
        end   = matRows[outIdx];
        for (iterator = start; iterator < end; iterator++) {
            prod += matData[matColStart[iterator] + outIdx] * vec[matCols[matColStart[iterator] + outIdx]];
        }
        out[matRowPerm[outIdx]] += prod;
    }
}

static void
spmvCSR(float *out, int *matCols, int *matRows, float *matData, float *vec, int dim)
{
    dim3 DimGrid(ceil((dim - 1) / 8.0), 1, 1);
    dim3 DimBlock(8, 1, 1);
    spmvCSRKernel<<<DimGrid, DimBlock>>>(out, matCols, matRows, matData, vec, dim);
}

static void
spmvJDS(float *out, int *matColStart, int *matCols, int *matRowPerm, int *matRows, float *matData, float *vec, int dim)
{
    dim3 DimGrid(ceil((dim - 1) / 8.0), 1, 1);
    dim3 DimBlock(8, 1, 1);

    spmvJDSKernel<<<DimGrid, DimBlock>>>(out, matColStart, matCols, matRowPerm, matRows, matData, vec, dim);
}

static void
sort(int *data, int *key, int start, int end)
{
    if ((end - start + 1) > 1) {
        int left = start, right = end;
        int pivot = key[right];
        while (left <= right) {
            while (key[left] > pivot) {
                left = left + 1;
            }
            while (key[right] < pivot) {
                right = right - 1;
            }
            if (left <= right) {
                int tmp     = key[left];
                key[left]   = key[right];
                key[right]  = tmp;
                tmp         = data[left];
                data[left]  = data[right];
                data[right] = tmp;
                left        = left + 1;
                right       = right - 1;
            }
        }
        sort(data, key, start, right);
        sort(data, key, left, end);
    }
}

void
CSRToJDS(int dim, int *csrRowPtr, int *csrColIdx, float *csrData, int **jdsRowPerm, int **jdsRowNNZ,
         int **jdsColStartIdx, int **jdsColIdx, float **jdsData)
{
    // Row Permutation Vector
    *jdsRowPerm = (int *)malloc(sizeof(int) * dim);
    for (int rowIdx = 0; rowIdx < dim; ++rowIdx) {
        (*jdsRowPerm)[rowIdx] = rowIdx;
    }

    // Number of non-zeros per row
    *jdsRowNNZ = (int *)malloc(sizeof(int) * dim);
    for (int rowIdx = 0; rowIdx < dim; ++rowIdx) {
        (*jdsRowNNZ)[rowIdx] = csrRowPtr[rowIdx + 1] - csrRowPtr[rowIdx];
    }

    // Sort rows by number of non-zeros
    sort(*jdsRowPerm, *jdsRowNNZ, 0, dim - 1);

    // Starting point of each compressed column
    int maxRowNNZ = (*jdsRowNNZ)[0]; // Largest number of non-zeros per row
    printf("jdsRowNNZ = %d\n", maxRowNNZ);
    *jdsColStartIdx      = (int *)malloc(sizeof(int) * maxRowNNZ);
    (*jdsColStartIdx)[0] = 0; // First column starts at 0
    for (int col = 0; col < maxRowNNZ - 1; ++col) {
        // Count the number of rows with entries in this column
        int count = 0;
        for (int idx = 0; idx < dim; ++idx) {
            if ((*jdsRowNNZ)[idx] > col) {
                ++count;
            }
        }
        (*jdsColStartIdx)[col + 1] = (*jdsColStartIdx)[col] + count;
    }

    // Sort the column indexes and data
    const int NNZ = csrRowPtr[dim];
    printf("NNZ = %d\n", NNZ);
    *jdsColIdx = (int *)malloc(sizeof(int) * NNZ);
    printf("dim = %d\n", dim);
    *jdsData = (float *)malloc(sizeof(float) * NNZ);
    for (int idx = 0; idx < dim; ++idx) { // For every row
        int row    = (*jdsRowPerm)[idx];
        int rowNNZ = (*jdsRowNNZ)[idx];
        for (int nnzIdx = 0; nnzIdx < rowNNZ; ++nnzIdx) {
            int jdsPos           = (*jdsColStartIdx)[nnzIdx] + idx;
            int csrPos           = csrRowPtr[row] + nnzIdx;
            (*jdsColIdx)[jdsPos] = csrColIdx[csrPos];
            (*jdsData)[jdsPos]   = csrData[csrPos];
        }
    }
}

int
main(int argc, char **argv)
{
    bool usingJDSQ;
    int *hostCSRCols;
    int *hostCSRRows;
    float *hostCSRData;
    int *hostJDSColStart;
    int *hostJDSCols;
    int *hostJDSRowPerm;
    int *hostJDSRows;
    float *hostJDSData;
    float *hostVector;
    float *hostOutput;
    int *deviceCSRCols;
    int *deviceCSRRows;
    float *deviceCSRData;
    int *deviceJDSColStart;
    int *deviceJDSCols;
    int *deviceJDSRowPerm;
    int *deviceJDSRows;
    float *deviceJDSData;
    float *deviceVector;
    float *deviceOutput;
    int dim, ncols, nrows, ndata;
    int maxRowNNZ;

    printf("Dataset Folder %s\n", argv[1]);
    printf("Importing data and creating memory on host\n");
    char filename[80];
    sprintf(filename, "%smode.flag", argv[1]);
    printf("%s\n", filename);
    FILE *fmode = fopen(filename, "r");
    fscanf(fmode, "%d/n", &usingJDSQ);
    fclose(fmode);
    sprintf(filename, "%scol.raw", argv[1]);
    printf("%s\n", filename);
    FILE *fcol = fopen(filename, "r");
    fscanf(fcol, "%d\n", &ncols);
    hostCSRCols = new int[ncols];
    for (int n = 0; n < ncols; n++)
        fscanf(fcol, "%d\n", hostCSRCols + n);
    fclose(fcol);
    sprintf(filename, "%srow.raw", argv[1]);
    printf("%s\n", filename);
    FILE *frow = fopen(filename, "r");
    fscanf(frow, "%d\n", &nrows);
    hostCSRRows = new int[nrows];
    for (int n = 0; n < nrows; n++)
        fscanf(frow, "%d\n", hostCSRRows + n);
    fclose(frow);
    sprintf(filename, "%sdata.raw", argv[1]);
    printf("%s\n", filename);
    FILE *fdata = fopen(filename, "r");
    fscanf(fdata, "%d\n", &ndata);
    hostCSRData = new float[ndata];
    for (int n = 0; n < ndata; n++)
        fscanf(fdata, "%e\n", hostCSRData + n);
    fclose(fdata);
    sprintf(filename, "%svec.raw", argv[1]);
    printf("%s\n", filename);
    FILE *fvec = fopen(filename, "r");
    fscanf(fvec, "%d\n", &dim);
    hostVector = new float[dim];
    for (int n = 0; n < dim; n++)
        fscanf(fvec, "%e\n", hostVector + n);
    fclose(fvec);
    if (usingJDSQ)
        printf("JDS Multiplication\n");
    else
        printf("CSR Multiplication\n");
    printf("#Columns: %d, #Rows: %d, #Data: %d, Dim: %d\n", ncols, nrows, ndata, dim);

    /*printf("hostCSRRows\n");
    for(int n = 0; n < nrows; n++)
      printf("%d\n",hostCSRRows[n]);
    printf("hostCSRCols\n");
    for(int n = 0; n < ncols; n++)
      printf("%d\n",hostCSRCols[n]);
    printf("hostVector\n");
    for(int n = 0; n < dim; n++)
      printf("%e\n",hostVector[n]);

    float *mat = new float[dim*dim];
    for(int n = 0; n < dim*dim; n++)
      mat[n] = 0;
    for(int m = 0; m < dim; m++)
      for(int n = hostCSRRows[m]; n < hostCSRRows[m+1]; n++)
        mat[m*dim+hostCSRCols[n]] = hostCSRData[n];
    printf("\n");
    for(int m = 0; m < dim; m++){
      for(int n = 0; n < dim; n++)
        printf("%.0f ",mat[m*dim+n]);
      printf("\n");
    }*/

    hostOutput = (float *)malloc(sizeof(float) * dim);

    if (usingJDSQ) {
        CSRToJDS(dim, hostCSRRows, hostCSRCols, hostCSRData, &hostJDSRowPerm, &hostJDSRows, &hostJDSColStart,
                 &hostJDSCols, &hostJDSData);
        maxRowNNZ = hostJDSRows[0];
    }

    printf("Allocating GPU memory.\n");
    if (usingJDSQ) {
        hipMalloc((void **)&deviceJDSColStart, sizeof(int) * maxRowNNZ);
        hipMalloc((void **)&deviceJDSCols, sizeof(int) * ndata);
        hipMalloc((void **)&deviceJDSRowPerm, sizeof(int) * dim);
        hipMalloc((void **)&deviceJDSRows, sizeof(int) * dim);
        hipMalloc((void **)&deviceJDSData, sizeof(float) * ndata);
    } else {
        hipMalloc((void **)&deviceCSRCols, sizeof(int) * ncols);
        hipMalloc((void **)&deviceCSRRows, sizeof(int) * nrows);
        hipMalloc((void **)&deviceCSRData, sizeof(float) * ndata);
    }
    hipMalloc((void **)&deviceVector, sizeof(float) * dim);
    hipMalloc((void **)&deviceOutput, sizeof(float) * dim);

    printf("Copying input memory to the GPU.\n");
    if (usingJDSQ) {
        hipMemcpy(deviceJDSColStart, hostJDSColStart, sizeof(int) * maxRowNNZ, hipMemcpyHostToDevice);
        hipMemcpy(deviceJDSCols, hostJDSCols, sizeof(int) * ndata, hipMemcpyHostToDevice);
        hipMemcpy(deviceJDSRowPerm, hostJDSRowPerm, sizeof(int) * dim, hipMemcpyHostToDevice);
        hipMemcpy(deviceJDSRows, hostJDSRows, sizeof(int) * dim, hipMemcpyHostToDevice);
        hipMemcpy(deviceJDSData, hostJDSData, sizeof(float) * ndata, hipMemcpyHostToDevice);
    } else {
        hipMemcpy(deviceCSRCols, hostCSRCols, sizeof(int) * ncols, hipMemcpyHostToDevice);
        hipMemcpy(deviceCSRRows, hostCSRRows, sizeof(int) * nrows, hipMemcpyHostToDevice);
        hipMemcpy(deviceCSRData, hostCSRData, sizeof(float) * ndata, hipMemcpyHostToDevice);
    }
    hipMemcpy(deviceVector, hostVector, sizeof(float) * dim, hipMemcpyHostToDevice);

    typedef std::chrono::high_resolution_clock Clock;
    typedef std::chrono::duration<double> Duration;
    printf("Performing CUDA computation\n");
    auto start = Clock::now();
    if (usingJDSQ) {
        spmvJDS(deviceOutput, deviceJDSColStart, deviceJDSCols, deviceJDSRowPerm, deviceJDSRows, deviceJDSData,
                deviceVector, dim);
    } else {
        spmvCSR(deviceOutput, deviceCSRCols, deviceCSRRows, deviceCSRData, deviceVector, dim);
    }
    hipDeviceSynchronize();
    Duration elapsed = Clock::now() - start;
    std::cout << elapsed.count() << " seconds\n";

    printf("Copying output memory to the CPU\n");
    hipMemcpy(hostOutput, deviceOutput, sizeof(float) * dim, hipMemcpyDeviceToHost);

    printf("Freeing GPU Memory\n");
    hipFree(deviceCSRCols);
    hipFree(deviceCSRRows);
    hipFree(deviceCSRData);
    hipFree(deviceVector);
    hipFree(deviceOutput);
    if (usingJDSQ) {
        hipFree(deviceJDSColStart);
        hipFree(deviceJDSCols);
        hipFree(deviceJDSRowPerm);
        hipFree(deviceJDSRows);
        hipFree(deviceJDSData);
    }

    sprintf(filename, "%soutput.raw", argv[1]);
    printf("%s\n", filename);
    FILE *fout = fopen(filename, "r");
    fscanf(fout, "%d\n", &dim);
    printf("dim: %d\n", dim);
    float *hostOut = new float[dim];
    for (int n = 0; n < dim; n++)
        fscanf(fout, "%e\n", &hostOut[n]);
    fclose(fout);

    bool res = true;
    for (int n = 0; n < dim; n++)
        if (hostOutput[n] != hostOut[n]) {
            printf("%e %e does not match at row %d!\n", hostOutput[n], hostOut[n], n);
            res = false;
        }
    if (res)
        printf("Result is correct!\n");
    else
        printf("some elements do not match\n");

    free(hostCSRCols);
    free(hostCSRRows);
    free(hostCSRData);
    free(hostVector);
    free(hostOutput);
    if (usingJDSQ) {
        free(hostJDSColStart);
        free(hostJDSCols);
        free(hostJDSRowPerm);
        free(hostJDSRows);
        free(hostJDSData);
    }

    return 0;
}
