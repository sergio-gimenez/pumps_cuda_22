#include "hip/hip_runtime.h"
#include "kernel.h"

#include <stdio.h>

__global__ void axpy_cuda(double *x, double *y, double alpha, long N)
{
   long int i = blockIdx.x * blockDim.x + threadIdx.x;
   if(i < N) y[i] *= alpha * x[i];
}
