#include "hip/hip_runtime.h"
#include <cstdint>
#include "helper.hpp"

__global__ void
s2g_gpu_gather_kernel(uint32_t *in, uint32_t *out, int len)
{
    // Nice reference: https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#shared-memory

    int outIdx = blockIdx.x * blockDim.x + threadIdx.x; // Indice que itera en el array de entrada

    if (outIdx < len) {
        int out_reg = 0; // Allocate out register in stack
        int inIdx;       // Indice que se recorre el vector de entrada
        for (inIdx = 0; inIdx < len; ++inIdx) {
            int intermediate = outInvariant(in[inIdx]);
            out_reg += outDependent(intermediate, inIdx, outIdx);
        }
        out[outIdx] += out_reg;
    }
}

static void
s2g_cpu_gather(uint32_t *in, uint32_t *out, int len)
{
    for (int outIdx = 0; outIdx < len; ++outIdx) {  // Recorre las salidas
        int out_reg = 0;                            // Valor del registro de salida
        for (int inIdx = 0; inIdx < len; ++inIdx) { // Recorre las entredas para cada salida
            int intermediate =
                outInvariant(in[inIdx]); // Devuelve el valor de la variable en la posicion inIdx en el array in.
            out_reg += outDependent(intermediate, inIdx,
                                    outIdx); // Suma el valor obtenido de realizar la operacion outDependent (calcula un
                                             // valor que depende de los componentes de in, arbitrario).
        }
        out[outIdx] += out_reg; // Escribe en la posicion del array out, el valor obtenido que dependia del vector in.
    }
}

static void
s2g_gpu_gather(uint32_t *in, uint32_t *out, int len)
{
    // Nice reference:
    // https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#device-memory

    // "in" and "out" passed by reference, assume already in host memory and initialized
    // What is being passed here, in reality is the cuda memory allocation.

    // Invoke kernel
    int threadsPerBlock = 256; // Arbitrary number?

    int blocksPerGrid = (len + threadsPerBlock - 1) / threadsPerBlock;
    s2g_gpu_gather_kernel<<<blocksPerGrid, threadsPerBlock>>>(in, out, len);
}

static int
eval(int inputLength)
{
    uint32_t *deviceInput  = nullptr;
    uint32_t *deviceOutput = nullptr;

    const std::string conf_info = std::string("gather[len:") + std::to_string(inputLength) + "]";
    INFO("Running " << conf_info);

    auto hostInput = generate_input(inputLength);

    const size_t byteCount = inputLength * sizeof(uint32_t);

    timer_start("Allocating GPU memory.");
    THROW_IF_ERROR(hipMalloc((void **)&deviceInput, byteCount));
    THROW_IF_ERROR(hipMalloc((void **)&deviceOutput, byteCount));
    timer_stop();

    timer_start("Copying input memory to the GPU.");
    THROW_IF_ERROR(hipMemcpy(deviceInput, hostInput.data(), byteCount, hipMemcpyHostToDevice));
    THROW_IF_ERROR(hipMemset(deviceOutput, 0, byteCount));
    timer_stop();

    //////////////////////////////////////////
    // GPU Gather Computation
    //////////////////////////////////////////
    timer_start("Performing GPU Gather computation");
    s2g_gpu_gather(deviceInput, deviceOutput, inputLength);
    timer_stop();

    std::vector<uint32_t> hostOutput(inputLength);

    timer_start("Copying output memory to the CPU");
    THROW_IF_ERROR(hipMemcpy(hostOutput.data(), deviceOutput, byteCount, hipMemcpyDeviceToHost));
    timer_stop();

    auto expected = compute_output(hostInput, inputLength);
    verify(expected, hostOutput);

    hipFree(deviceInput);
    hipFree(deviceOutput);

    return 0;
}

TEST_CASE("Gather", "[gather]")
{
    SECTION("[inputSize:1024]") { eval(1024); }
    SECTION("[inputSize:2048]") { eval(2048); }
    SECTION("[inputSize:2047]") { eval(2047); }
    SECTION("[inputSize:2049]") { eval(2049); }
    SECTION("[inputSize:9101]") { eval(9101); }
    SECTION("[inputSize:9910]") { eval(9910); }
    SECTION("[inputSize:8192]") { eval(8192); }
    SECTION("[inputSize:8193]") { eval(8193); }
    SECTION("[inputSize:8191]") { eval(8191); }
    SECTION("[inputSize:16191]") { eval(16191); }
}
