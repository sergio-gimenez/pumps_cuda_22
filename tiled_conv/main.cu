#include "hip/hip_runtime.h"
#include "helper.hpp"

// Sequential code for the forward path of the convolution layer
// You should not modify this code
static void
conv_forward_valid(const float *X, const shape &xdims, const float *W, const shape &wdims, float *Y, const shape &ydims)
{
    std::fill(Y, Y + ydims.flattened_length(), 0);

    for (auto i : range(0, ydims.num)) {
        for (auto m : range(0, ydims.depth)) {      // for each output feature map
            for (auto h : range(0, ydims.height)) { // for each output element
                for (auto w : range(0, ydims.width)) {
                    const auto yoffset = ((i * ydims.depth + m) * ydims.height + h) * ydims.width + w;
                    for (auto c : range(0, xdims.depth)) {         // sum over all input feature maps
                        for (auto p : range(0, wdims.height)) {    // filter height
                            for (auto q : range(0, wdims.width)) { // filter width
                                const auto xoffset =
                                    ((((i * xdims.depth) + c) * xdims.height) + (h + p)) * xdims.width + (w + q);
                                const auto woffset = ((((m * wdims.depth) + c) * wdims.height) + p) * wdims.width + q;
                                Y[yoffset] += X[xoffset] * W[woffset];
                            }
                        }
                    }
                }
            }
        }
    }
}

// Baseline GPU kernel code for forward convolution.
// One thread per output index
// You should not modify this kernel as it is used for correctness comparison.
// Instead, define a new one below
__global__ void
conv_forward_baseline_kernel(const float *X, const shape xdims, const float *W, const shape wdims, float *Y,
                             const shape ydims)
{
    const size_t gx = blockIdx.x * blockDim.x + threadIdx.x;
    for (size_t i = gx; i < ydims.num * ydims.depth * ydims.height * ydims.width; i += blockDim.x * gridDim.x) {
        Y[i] = 0.f;
    }

    for (size_t i = gx; i < ydims.num; i += gridDim.x * blockDim.x) {
        for (auto m : range(0, ydims.depth)) {      // for each output feature map
            for (auto h : range(0, ydims.height)) { // for each output element
                for (auto w : range(0, ydims.width)) {
                    const size_t yoffset = ((i * ydims.depth + m) * ydims.height + h) * ydims.width + w;
                    for (auto c : range(0, xdims.depth)) {         // sum over all input feature maps
                        for (auto p : range(0, wdims.height)) {    // filter height
                            for (auto q : range(0, wdims.width)) { // filter width
                                const size_t xoffset =
                                    ((((i * xdims.depth) + c) * xdims.height) + (h + p)) * xdims.width + (w + q);
                                const size_t woffset = ((((m * wdims.depth) + c) * wdims.height) + p) * wdims.width + q;
                                Y[yoffset] += X[xoffset] * W[woffset];
                            }
                        }
                    }
                }
            }
        }
    }
}

// Host code to configure baseline GPU kernel
static void
convlayer_gpu_baseline(const float *X, const shape &xdims, const float *W, const shape &wdims, float *Y,
                       const shape &ydims)
{
    dim3 dimGrid(1);
    dim3 dimBlock(32);

    conv_forward_baseline_kernel<<<dimGrid, dimBlock>>>(X, xdims, W, wdims, Y, ydims);
    THROW_IF_ERROR(hipGetLastError());
}

static void
unroll(int C, int H, int W, int K, float *X, float *X_unroll)
{
    int H_out = H - K + 1;
    int W_out = W - K + 1;
    for (int c = 0; c < C; c++) {
        // beginning row index of the section for channel C input feature
        // map in the unrolled matrix
        int w_base = c * (K * K);
        for (int p = 0; p < K; p++) {
            for (int q = 0; q < K; q++) {
                int h_unroll = w_base + p * K + q;
                for (int h = 0; h < H_out; h++) {
                    for (int w = 0; w < W_out; w++) {
                        int w_unroll                 = h * W_out + w;
                        X_unroll[h_unroll, w_unroll] = X(c, h + p, w + q);
                    }
                }
            }
        }
    }
}

// Implement your optimized kernel here.
// Don't forget to modify the host code below, if needed!
__global__ void
conv_forward_opt_kernel(const float *X, const shape xdims, const float *W, const shape wdims, float *Y,
                        const shape ydims)
{
    // 1. The kernel that implements a convolution layer performs a tiled matrix multiplication on the conceptual
    // unrolled input matrix.

    // 2. When loading each tile from the “unrolled input matrix”, the kernel extracts the tile elements from the
    // original input feature maps.

    // 3. The kernel loads the filter weights from the global memory into the shared memory.

    // 4. The kernel performs the matrix multiplication on the tile and the filter weights.

    // 5. The kernel stores the result of the matrix multiplication in the global memory.

    
    


}

// Host code to configure baseline GPU kernel
static void
convlayer_gpu_opt(const float *X, const shape &xdims, const float *W, const shape &wdims, float *Y, const shape &ydims)
{
    //@@ YOUR CODE HERE
    // dim3 dimGrid(...);
    // dim3 dimBlock(...);
    conv_forward_opt_kernel<<<dimGrid, dimBlock>>>(X, xdims, W, wdims, Y, ydims);
    THROW_IF_ERROR(hipGetLastError());
}

static int
eval(const shape wDims, const shape xDims)
{
    // Generate model
    const auto conf_info = std::string("conv[wDims:") + std::to_string(wDims.num) + "," + std::to_string(wDims.depth) +
                           "," + std::to_string(wDims.height) + "," + std::to_string(wDims.width) +
                           " xDims:" + std::to_string(xDims.num) + "," + std::to_string(xDims.depth) + "," +
                           std::to_string(xDims.height) + "," + std::to_string(xDims.width) + "]";
    INFO("Running " << conf_info);

    // Generate convolution weights
    float *hostW = allocate<float>(wDims);
    generate_convfilters(hostW, wDims);

    // generate input feature map
    float *hostX = allocate<float>(xDims);
    generate_data(hostX, xDims);

    // generate output feature map for verification
    const shape ydims = {xDims.num, wDims.num, (xDims.height - wDims.height + 1), (xDims.width - wDims.width + 1)};
    INFO("Allocating output tensor [" << ydims.num << "," << ydims.depth << "," << ydims.height << "," << ydims.width
                                      << "]");
    float *hostY    = allocate<float>(ydims);
    float *expected = allocate<float>(ydims);
    generate_data(hostY, ydims);

    const size_t wByteCount = wDims.flattened_length() * sizeof(float);
    const size_t xByteCount = xDims.flattened_length() * sizeof(float);
    const size_t yByteCount = ydims.flattened_length() * sizeof(float);

    float *deviceW = nullptr, *deviceX = nullptr, *deviceY = nullptr;
    timer_start("Allocating GPU memory.");
    THROW_IF_ERROR(hipMalloc((void **)&deviceW, wByteCount));
    THROW_IF_ERROR(hipMalloc((void **)&deviceX, xByteCount));
    THROW_IF_ERROR(hipMalloc((void **)&deviceY, yByteCount));
    timer_stop();

    timer_start("Copying inputs to the GPU.");
    THROW_IF_ERROR(hipMemcpy(deviceW, hostW, wByteCount, hipMemcpyDefault));
    THROW_IF_ERROR(hipMemcpy(deviceX, hostX, xByteCount, hipMemcpyDefault));
    timer_stop();

    //////////////////////////////////////////
    // GPU Gather Computation
    //////////////////////////////////////////
    timer_start("Performing GPU convlayer");
    convlayer_gpu_opt(deviceX, xDims, deviceW, wDims, deviceY, ydims);
    THROW_IF_ERROR(hipDeviceSynchronize());
    timer_stop();

    timer_start("Copying output to the CPU");
    THROW_IF_ERROR(hipMemcpy(hostY, deviceY, yByteCount, hipMemcpyDefault));
    timer_stop();

    // verify with provided implementation
    convlayer_gpu_baseline(deviceX, xDims, deviceW, wDims, deviceY, ydims);
    THROW_IF_ERROR(hipDeviceSynchronize());
    THROW_IF_ERROR(hipMemcpy(expected, deviceY, yByteCount, hipMemcpyDefault));
    verify(expected, hostY, ydims);

    THROW_IF_ERROR(hipFree(deviceW));
    THROW_IF_ERROR(hipFree(deviceX));
    THROW_IF_ERROR(hipFree(deviceY));
    free(hostW);
    free(hostX);
    free(hostY);
    free(expected);

    return 0;
}

TEST_CASE("Convlayer", "[convlayer]")
{
    SECTION("[wDims:0,0,0,0 xDims:100,1,32,32]") { eval({0, 0, 0, 0}, {100, 1, 32, 32}); }
    SECTION("[wDims:1,1,1,1 xDims:100,1,32,32]") { eval({1, 1, 1, 1}, {100, 1, 32, 32}); }
    SECTION("[wDims:32,1,5,5 xDims:1000,1,28,28]") { eval({32, 1, 5, 5}, {1000, 1, 28, 28}); }
    SECTION("[wDims:16,1,3,3 xDims:100,1,32,32]") { eval({16, 1, 3, 3}, {100, 1, 32, 32}); }
}
